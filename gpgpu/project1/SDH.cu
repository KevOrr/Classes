/* ==================================================================
   Programmer: Yicheng Tu (ytu@cse.usf.edu)
   The basic SDH algorithm implementation for 3D data
   To compile: nvcc SDH.c -o SDH in the C4 lab machines
   ==================================================================
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <sys/time.h>


const long BOX_SIZE = 23000; /* size of the data box on one dimension */

/* descriptors for single atom in the tree */
typedef struct atomdesc {
    double x_pos;
    double y_pos;
    double z_pos;
} atom;

typedef struct hist_entry {
    //float min;
    //float max;
    unsigned long long d_cnt;   /* need a long long type as the count might be huge */
} bucket;

struct debuginfo {
    int idx;
    int ran;
    int i;
    int j;
    float dist;
    int which_bucket;
};


bucket * histogram;  /* list of all buckets in the histogram   */
unsigned long long PDH_acnt; /* total number of data points            */
int num_buckets;  /* total number of buckets in the histogram */
double PDH_res;  /* value of w                             */
atom * atom_list;  /* list of all data points                */

/* These are for an old way of tracking time */
struct timezone Idunno;
struct timeval startTime, endTime;


/*
 distance of two points in the atom_list
*/
double p2p_distance(int ind1, int ind2) {

    double x1 = atom_list[ind1].x_pos;
    double x2 = atom_list[ind2].x_pos;
    double y1 = atom_list[ind1].y_pos;
    double y2 = atom_list[ind2].y_pos;
    double z1 = atom_list[ind1].z_pos;
    double z2 = atom_list[ind2].z_pos;

    return sqrt((x1 - x2)*(x1-x2) + (y1 - y2)*(y1 - y2) + (z1 - z2)*(z1 - z2));
}


/*
 brute-force SDH solution in a single CPU thread
*/
int PDH_baseline() {
    int i, j, h_pos;
    double dist;

    for(i = 0; i < PDH_acnt; i++) {
        for(j = i+1; j < PDH_acnt; j++) {
            dist = p2p_distance(i,j);
            h_pos = (int) (dist / PDH_res);
            if (h_pos >= 0 && h_pos < num_buckets)
                histogram[h_pos].d_cnt++;
            else
                printf("Warning: value %lf falls outside histogram", dist);
        }
    }
    return 0;
}

__global__
void PDH_kernel(long n_threads, bucket *d_buckets, int n_buckets, const atom *d_atoms, double w
#ifdef DEBUG
                , struct debuginfo *d_dinfo
#endif
    ) {
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx >= n_threads)
        return;
    
    // Please don't make me explain. It was 2am and I scribbled some math and it works
    int i = (sqrt(8.0*idx + 1.0) - 1.0)/2;
    int j = idx - i*(i + 1)/2;
    i++;

    double deltax = d_atoms[i].x_pos - d_atoms[j].x_pos;
    double deltay = d_atoms[i].y_pos - d_atoms[j].y_pos;
    double deltaz = d_atoms[i].z_pos - d_atoms[j].z_pos;

    double dist = sqrt(deltax*deltax + deltay*deltay + deltaz*deltaz);
    int h_pos = (int) (dist / w);
    if (h_pos >= 0 && h_pos < n_buckets)
        // atomicAdd(&d_buckets[h_pos].d_cnt, 1);
        d_buckets[h_pos*n_threads + idx].d_cnt++; // Coalesce!



#ifdef DEBUG
    d_dinfo[idx].idx = idx;
    d_dinfo[idx].i = i;
    d_dinfo[idx].j = j;
    d_dinfo[idx].ran = 1;
    d_dinfo[idx].dist = dist;
    d_dinfo[idx].which_bucket = (int) (dist / w);
#endif
}

void PDH_gpu() {
    unsigned long num_threads = PDH_acnt*(PDH_acnt - 1)/2;

    // allocate histogram
    bucket *d_buckets;
    hipMalloc(&d_buckets, sizeof(*histogram) * num_buckets);
    hipMemset(d_buckets, 0, sizeof(*histogram) * num_buckets);

#ifdef DEBUG
    // allocate debuginfo
    struct debuginfo *d_dinfo;
    cudaMalloc(&d_dinfo, sizeof(*d_dinfo) * num_threads);
    cudaMemset(d_dinfo, 0, sizeof(*d_dinfo) * num_threads);
#endif

    // Copy atoms to device
    atom *d_atoms;
    hipMalloc(&d_atoms, sizeof(*atom_list) * PDH_acnt);
    hipMemcpy(d_atoms, atom_list, sizeof(*atom_list) * PDH_acnt, hipMemcpyHostToDevice);

    PDH_kernel<<<(num_threads + 255)/256, 256>>>(num_threads, d_buckets, num_buckets, d_atoms, PDH_res
    // PDH_kernel<<<1, num_threads>>>(num_threads, d_buckets, num_buckets, d_atoms, PDH_res
#ifdef DEBUG
                                                 , d_dinfo
#endif
        );
    // cudaDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA ERROR: %s\n", hipGetErrorString(err));
        puts("This is probably due to a too-large block count");
    }

    // Copy histogram from device and cleanup
    hipFree(d_atoms);
    hipMemcpy(histogram, d_buckets, sizeof(*histogram) * num_buckets, hipMemcpyDeviceToHost);
    hipFree(d_buckets);

#ifdef DEBUG
    // Copy debuginfo from device and cleanup
    struct debuginfo *h_dinfo = (struct debuginfo *) malloc(sizeof(*h_dinfo) * num_threads);
    cudaMemcpy(h_dinfo, d_dinfo, sizeof(*h_dinfo) * num_threads, cudaMemcpyDeviceToHost);
    cudaFree(d_dinfo);

    for (unsigned long long i=0; i<num_threads; i++) {
        printf("%llu: idx=%d, ran=%d, i=%d, j=%d, dist=%f, bucket=%d\n",
               i, h_dinfo[i].idx, h_dinfo[i].ran, h_dinfo[i].i, h_dinfo[i].j,
               h_dinfo[i].dist, h_dinfo[i].which_bucket);
    }
#endif
}


/*
 set a checkpoint and show the (natural) running time in seconds
*/
double report_running_time(const char *type) {
    long sec_diff, usec_diff;
    gettimeofday(&endTime, &Idunno);
    sec_diff = endTime.tv_sec - startTime.tv_sec;
    usec_diff= endTime.tv_usec - startTime.tv_usec;
    if (usec_diff < 0) {
        sec_diff--;
        usec_diff += 1000000;
    }
    printf("Running time for %s version: %ld.%06ld\n", type, sec_diff, usec_diff);
    return (double)(sec_diff*1.0 + usec_diff/1000000.0);
}


/*
 print the counts in all buckets of the histogram
*/
void output_histogram() {
    int i;
    unsigned long long total_cnt = 0;
    for(i=0; i<num_buckets; i++) {
        if (i%5 == 0) /* we print 5 buckets in a row */
            printf("\n%02d: ", i);
        printf("%15lld ", histogram[i].d_cnt);
        total_cnt += histogram[i].d_cnt;
        /* we also want to make sure the total distance count is correct */
        if (i == num_buckets - 1)
            printf("\n T:%lld \n", total_cnt);
        else printf("| ");
    }
}


int main(int argc, char **argv)
{
    int i;

    PDH_acnt = atoi(argv[1]);
    PDH_res = atof(argv[2]);
    // printf("args are %d and %f\n", PDH_acnt, PDH_res);

    num_buckets = (int)(BOX_SIZE * 1.732 / PDH_res) + 1;
    histogram = (bucket *)malloc(sizeof(bucket)*num_buckets);

    atom_list = (atom *)malloc(sizeof(atom)*PDH_acnt);


    srand(1);
    /* generate data following a uniform distribution */
    for(i = 0; i < PDH_acnt; i++) {
        atom_list[i].x_pos = ((double)(rand()) / RAND_MAX) * BOX_SIZE;
        atom_list[i].y_pos = ((double)(rand()) / RAND_MAX) * BOX_SIZE;
        atom_list[i].z_pos = ((double)(rand()) / RAND_MAX) * BOX_SIZE;
    }

    // CPU implementation
    puts("Running CPU version...");
    memset(histogram, 0, sizeof(*histogram) * num_buckets);
    gettimeofday(&startTime, &Idunno);
    PDH_baseline();
    report_running_time("CPU");
    output_histogram();

    // GPU implementation
    puts("\nRunning GPU version...");
    memset(histogram, 0, sizeof(*histogram) * num_buckets);
    gettimeofday(&startTime, &Idunno);
    PDH_gpu();
    report_running_time("GPU");
    output_histogram();


    return 0;
}
