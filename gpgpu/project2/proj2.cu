
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <errno.h>
#include <error.h>
#include <unistd.h>

const long BOX_SIZE = 23000; /* size of the data box on one dimension */
#define BLOCK_SIZE 512

#define EXITERROR() error_at_line(errno, errno, __FILE__, __LINE__, "pid %llu", (long long unsigned)getpid())

const char *argv0;

typedef struct hist_entry {
    //float min;
    //float max;
    unsigned long long d_cnt;   /* need a long long type as the count might be huge */
} bucket;

struct debuginfo {
    int idx;
    int ran;
    int i;
    int j;
    float dist;
    int which_bucket;
};


bucket * histogram;  /* list of all buckets in the histogram   */
unsigned long long num_points; /* total number of data points            */
int num_buckets;  /* total number of buckets in the histogram */
double resolution;  /* value of w                             */
double4 *h_points;


inline __device__ double dist3(double4 a, double4 b) {
    return norm3d(a.x - b.x, a.y - b.y, a.z - b.z);
}


__global__
void PDH_kernel(bucket *g_bins, size_t n_bins, double4 *g_points, size_t n_points, double res) {
    __shared__ double4 s_block[BLOCK_SIZE]; // R for inter-block, L for intra-block

    unsigned int t = threadIdx.x;
    unsigned int b = blockIdx.x;
    unsigned int B = blockDim.x;
    unsigned int M = gridDim.x;

    // Is our anchor point past the end of g_points?
    if (b*B + t >= n_points)
        return;

    // Anchor point in L
    double4 l_point = g_points[b*B + t];

    // Inter-block pairs, with block-level load balancing
    for (unsigned int i=1; i<=M/2; i++) {
        // Load balancing edge case
        if (M%2 == 0 && i >= M/2 && b >= M/2)
            continue;

        // Get R block
        unsigned int r = (b + i) % M;
        __syncthreads();
        if (r*B + t < n_points)
            s_block[t] = g_points[r*B + t];
        __syncthreads();

        // inter-block pairs
        for (unsigned int j=0; j<B; j++) {
            if (r*B + j < n_points) {
                double d = dist3(l_point, s_block[j]);
                unsigned long long bin = (unsigned long long)(d / res);
                if (bin <= n_bins)
                    atomicAdd(&g_bins[bin].d_cnt, 1);
            }
        }
    }

    // Get L block
    __syncthreads();
    if (b*B + t < n_points)
        s_block[t] = g_points[b*B + t];
    __syncthreads();

    // intra-block pairs
    for (unsigned int i=t+1; i<B && b*B+i<n_points; i++) {
        double d = dist3(s_block[t], s_block[i]);
        unsigned long long bin = (unsigned long long)(d / res);
        if (bin <= n_bins)
            atomicAdd(&g_bins[bin].d_cnt, 1);
    }
}

void PDH_gpu() {
    unsigned long num_threads = num_points*(num_points - 1)/2;

    // allocate histogram
    bucket *d_buckets;
    hipMalloc(&d_buckets, sizeof(*histogram) * num_buckets);
    hipMemset(d_buckets, 0, sizeof(*histogram) * num_buckets);

    // Copy points to device
    double4 *d_points;
    hipMalloc(&d_points, sizeof(*h_points) * num_points);
    hipMemcpy(d_points, h_points, sizeof(*h_points) * num_points, hipMemcpyHostToDevice);

    PDH_kernel<<<(num_points + BLOCK_SIZE - 1)/BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE>>>(
        d_buckets, num_buckets, d_points, num_points, resolution);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA ERROR: %s\n", hipGetErrorString(err));
        puts("This is probably due to a too-large block count");
    }

    // Copy histogram from device and cleanup
    hipFree(d_points);
    hipMemcpy(histogram, d_buckets, sizeof(*histogram) * num_buckets, hipMemcpyDeviceToHost);
    hipFree(d_buckets);
}


/*
 print the counts in all buckets of the histogram
*/
void output_histogram() {
    int i;
    unsigned long long total_cnt = 0;
    for(i=0; i<num_buckets; i++) {
        if (i%5 == 0) /* we print 5 buckets in a row */
            printf("\n%02d: ", i);
        printf("%15lld ", histogram[i].d_cnt);
        total_cnt += histogram[i].d_cnt;
        /* we also want to make sure the total distance count is correct */
        if (i == num_buckets - 1) {
            printf("\n          Total: %lld", total_cnt);
            printf("\n Expected total: %lld \n", num_points*(num_points - 1)/2);
        } else {
            printf("| ");
        }
    }
}

void usage(FILE *f, int ret) {
    fprintf(f, "USAGE: %s <SAMPLES> <RESOLUTION>\n", argv0);
    exit(ret);
}

int main(int argc, char **argv) {
    argv0 = argv[0];

    if (argc < 3)
        usage(stderr, 1);

    if (!strcmp(argv[1], "--help") || !strcmp(argv[1], "-h"))
        usage(stdout, 0);

    errno = 0;
    num_points = strtoull(argv[1], NULL, 10);
    if (errno != 0)
        EXITERROR();

    errno = 0;
    resolution = strtof(argv[2], NULL);
    if (errno != 0)
        EXITERROR();


    num_buckets = (int)(BOX_SIZE * 1.732 / resolution) + 1;
    histogram = (bucket *)malloc(sizeof(bucket)*num_buckets);

    h_points = (double4 *)malloc(sizeof(double4)*num_points);


    srand(1);
    /* generate data following a uniform distribution */
    for(size_t i = 0; i < num_points; i++) {
        h_points[i].x = ((double)(rand()) / RAND_MAX) * BOX_SIZE;
        h_points[i].y = ((double)(rand()) / RAND_MAX) * BOX_SIZE;
        h_points[i].z = ((double)(rand()) / RAND_MAX) * BOX_SIZE;
    }

    // GPU implementation
    puts("\nRunning Reg-SHM version...");
    memset(histogram, 0, sizeof(*histogram) * num_buckets);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    PDH_gpu();

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsed;
    hipEventElapsedTime(&elapsed, start, stop);
    printf("Running time for Reg-SHM version: %.6f sec\n", elapsed/1000.0);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    output_histogram();


    return 0;
}
